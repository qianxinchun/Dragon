#include "hip/hip_runtime.h"
#include "layers/vision/crop_layer.hpp"

//	a linear-mem copy kernel for the last two spatial axis
//	we re-implement it which is much efficient that caffe-master version
template <typename Dtype>
__global__ void	CopyKernel(const int n, const int height, const int width,
	const int src_outer_stride, const int src_inner_stride, const int dest_outer_stride, const int dest_inner_stride,
	const Dtype* src, Dtype* dest){
	CUDA_KERNEL_LOOP(idx, n){
		/*
		int src_start = idx / height*src_outer_stride
			+ idx%height*src_inner_stride;
		int dest_start = idx / height*dest_outer_stride
			+ idx%height*dest_inner_stride;*/
		int w = idx%width;
		int h = (idx / width) % height;
		int dest_idx = h*dest_inner_stride + w;
		int src_idx = h*src_inner_stride + w;
		dest[dest_idx] = src[src_idx];
	}
}


template <typename Dtype>
void CropLayer<Dtype>::copy_gpu(const vector<Blob<Dtype>*> &bottom, const vector<Blob<Dtype>*> &top,
	const vector<int>& offsets, vector<int> idxs, int cur_dim, const Dtype* src_data,
	Dtype* dest_data, bool is_forward){

	//	recursive-term
	if (cur_dim + 2 < top[0]->num_axes()){
		for (int i = 0; i < top[0]->shape(cur_dim); i++){
			//	store the pixel-idx of the current spatial axis
			idxs[cur_dim] = i;
			//	recursive for spatial axis
			copy_gpu(bottom, top, offsets, idxs, cur_dim + 1, src_data, dest_data, is_forward);
		}
	}
	//	terminal-term
	//	perform a linear-mem copy for the last two spatial axis
	//	you can also perform a last-n parallel algorithms for cuda kernel function
	else{
		const int lines = top[0]->shape(cur_dim);
		const int height = top[0]->shape(cur_dim);
		const int width= top[0]->shape(cur_dim+1);
		const int outer_num = height*width;
		vector<int> idx_off(cur_dim + 2, 0);
		for (int j = 0; j < cur_dim; j++) idx_off[j] = idxs[j] + offsets[j];
		idx_off[cur_dim] = offsets[cur_dim];
		idx_off[cur_dim+1] = offsets[cur_dim+1];
		const int src_outer_stride =
			bottom[0]->shape(cur_dim)*bottom[0]->shape(cur_dim + 1);
		const int src_inner_stride = bottom[0]->shape(cur_dim + 1);
		const int dest_outer_stride =
			top[0]->shape(cur_dim)*top[0]->shape(cur_dim + 1);
		const int dest_inner_stride = top[0]->shape(cur_dim + 1);
		//
		if (is_forward){
			const Dtype* bottom_data = bottom[0]->gpu_data() + bottom[0]->offset(idx_off);
			Dtype* top_data = top[0]->mutable_gpu_data() + top[0]->offset(idxs);
			CopyKernel<Dtype> << <GET_BLOCKS(outer_num), CUDA_NUM_THREADS >> >(
				outer_num, height, width, src_outer_stride, src_inner_stride,
				dest_outer_stride, dest_inner_stride,bottom_data, top_data);
		}else{
			const Dtype* top_diff = top[0]->gpu_diff() + top[0]->offset(idxs);
			Dtype* bottom_diff = bottom[0]->mutable_gpu_diff() + bottom[0]->offset(idx_off);
			CopyKernel<Dtype> << <GET_BLOCKS(outer_num), CUDA_NUM_THREADS >> >(
				outer_num, height, width, dest_outer_stride, dest_inner_stride,
				src_outer_stride, src_inner_stride, top_diff, bottom_diff);
		}
	}
}

template <typename Dtype>
void CropLayer<Dtype>::forward_gpu(const vector<Blob<Dtype>*> &bottom, const vector<Blob<Dtype>*> &top){
	vector<int> idxs(top[0]->num_axes(), 0);
	const Dtype* bottom_data = bottom[0]->gpu_data();
	Dtype* top_data = top[0]->mutable_gpu_data();
	copy_gpu(bottom, top, offsets, idxs, 0, bottom_data, top_data, true);
}

template <typename Dtype>
void CropLayer<Dtype>::backward_gpu(const vector<Blob<Dtype>*> &top,
	const vector<bool> &data_need_bp, const vector<Blob<Dtype>*> &bottom){
	if (!data_need_bp[0]) return;
	const Dtype* top_diff = top[0]->gpu_diff();
	Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
	//	must clear the last diff due to the different shape according mini-batches
	dragon_gpu_set(bottom[0]->count(), Dtype(0), bottom_diff);
	vector<int> idxs(top[0]->num_axes(), 0);
	copy_gpu(bottom, top, offsets, idxs, 0, top_diff, bottom_diff, false);
}


INSTANTIATE_LAYER_GPU_FUNCS(CropLayer);