#include "layers/vision/conv_layer.hpp"

template<typename Dtype>
void ConvolutionLayer<Dtype>::forward_gpu(const vector<Blob<Dtype>*> &bottom, const vector<Blob<Dtype>*> &top){
	//	4D(out_channels,in_channels,kernel_h,kernel_w)
	const Dtype* weights = blobs[0]->gpu_data();
	//	multi-input
	for (int i = 0; i < bottom.size(); i++){
		//	4D(batch_size,channels,height,width)
		const Dtype* bottom_data = bottom[i]->gpu_data();
		//	call reshape() to set a top blob referring to a bottom blob
		//	so top/bottom has the same blob quantity
		Dtype *top_data = top[i]->mutable_gpu_data();
		//	scan a batch
		for (int n = 0; n < num; n++){
			//	Wx
			forward_gpu_gemm(bottom_data + n*bottom_dim, weights, top_data + n*top_dim);
			if (bias_term){
				const Dtype* bias = blobs[1]->gpu_data();
				//	Wx+b
				forward_gpu_bias(top_data + n*top_dim, bias);
			}
		}
	}
}

template<typename Dtype>
void DeconvolutionLayer<Dtype>::forward_gpu(const vector<Blob<Dtype>*> &bottom, const vector<Blob<Dtype>*> &top){
	const Dtype* weights = blobs[0]->gpu_data();
	for (int i = 0; i < bottom.size(); i++){
		const Dtype* bottom_data = bottom[i]->gpu_data();
		Dtype *top_data = top[i]->mutable_gpu_data();
		for (int n = 0; n < num; n++){
			backward_gpu_gemm(bottom_data + n*bottom_dim, weights, top_data + n*top_dim);
			if (bias_term){
				const Dtype* bias = blobs[1]->gpu_data();
				forward_gpu_bias(top_data + n*top_dim, bias);
			}
		}
	}
}

template<typename Dtype>
void ConvolutionLayer<Dtype>::backward_gpu(const vector<Blob<Dtype>*> &top,
	const vector<bool> &data_need_bp, const vector<Blob<Dtype>*> &bottom){
	const Dtype* weights = blobs[0]->gpu_data();
	Dtype *weight_diff = blobs[0]->mutable_gpu_diff();
	//	multi-output
	//	we define sub-gradient as delta
	//	delta_(layer+1)=top->diff
	for (int i = 0; i < top.size(); i++){
		const Dtype* top_diff = top[i]->gpu_diff();
		const Dtype* bottom_data = bottom[i]->gpu_data();
		Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
		if (bias_term && param_need_bp[1]){
			Dtype *bias_diff = blobs[1]->mutable_gpu_diff();
			//	bias_diff += delta_(layer+1)
			//	a bias contributed to a channle's all spatial_dim
			//	we use gemv to combine the spatial_dim
			//	also we need sum up delta for all units in a batch
			for (int n = 0; n < num; n++)
				backward_gpu_bias(bias_diff, top_diff + n*top_dim);
		}
		if (param_need_bp[0] || data_need_bp[i]){
			for (int n = 0; n < num; n++){
				//	weight_diff += delta_(layer+1)*col
				//	in fully-connected layer it should be weight_diff+= delta*input
				//	we use im2col do a patch and extract the relevent input pixels in a col
				//	so in conv_layer, we replace input with col(patched input)
				//	also we need sum up delta for all units in a batch
				if (param_need_bp[0])
					weight_gpu_gemm(bottom_data + n*bottom_dim, top_diff + n*top_dim, weight_diff);
				if (data_need_bp[i])
					//	bottom_diff += delta_(layer+1)*weights
					//	bottom_diff actually is delta_(layer) and will be used in prev layer
					//	normally, bottom_diff += delta_(layer+1)*weights*f'(input)
					//	it skip the the grad of activative function
					//	we will add it in activative function layers
					backward_gpu_gemm(top_diff + n*top_dim, weights, bottom_diff + n*bottom_dim);
			}
		}
	}
}

template<typename Dtype>
void DeconvolutionLayer<Dtype>::backward_gpu(const vector<Blob<Dtype>*> &top,
	const vector<bool> &data_need_bp, const vector<Blob<Dtype>*> &bottom){
	const Dtype* weights = blobs[0]->gpu_data();
	Dtype *weight_diff = blobs[0]->mutable_gpu_diff();
	for (int i = 0; i < top.size(); i++){
		const Dtype* top_diff = top[i]->gpu_diff();
		const Dtype* bottom_data = bottom[i]->gpu_data();
		Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
		if (bias_term && param_need_bp[1]){
			Dtype *bias_diff = blobs[1]->mutable_gpu_diff();
			for (int n = 0; n < num; n++)
				backward_gpu_bias(bias_diff, top_diff + n*top_dim);
		}
		if (param_need_bp[0] || data_need_bp[i]){
			for (int n = 0; n < num; n++){
				if (param_need_bp[0])
					weight_gpu_gemm(top_diff + n*top_dim, bottom_data + n*bottom_dim, weight_diff);
				if (data_need_bp[i])
					//	note that we may compute im2col for top_diff in we weight_gemm
					//	skip im2col to speed up de-conv process
					forward_gpu_gemm(top_diff + n*top_dim, weights, bottom_diff + n*bottom_dim, param_need_bp[0]);
			}
		}
	}
}

INSTANTIATE_LAYER_GPU_FUNCS(ConvolutionLayer);
INSTANTIATE_LAYER_GPU_FUNCS(DeconvolutionLayer);